#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <math.h>
#include <algorithm>
#include <utility>
#include <cfloat>
#include <cmath>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


#include <thrust/pair.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#define ROWS 440
#define COLS 138672
#define K 5

using std::vector;
using std::string;
using std::ifstream;
using std::getline;
using std::cout;
using std::endl;


// input data to be read from file line by line needs to be split
vector<string> split(string s, char delimeter='\t') {
	vector<string> splitted;

	int num_elem = s.size();
	int i = 0;
	int j;
	while (i < num_elem) {
		j = s.find(delimeter, i);
		if (j == -1){
			splitted.push_back(s.substr(i));
			break;
		}
		splitted.push_back(s.substr(i,j-i));
		i = j+1;
	}
	return splitted;
}



// read line by line and store data in a 1d array
void readData1d(ifstream& in, thrust::host_vector<int>& data) {
	string d;
	int rows = (int)ROWS;
	for (int i = 0; i < rows; i++) {
		getline(in, d);
		data[i] = (int) atof(d.c_str());	
	}
	return ;
}


// read line by line and store data in a 2d array
void readData(ifstream& in, thrust::host_vector<float>& data, char delimeter = '\t') {
	string d;
	vector<string> dSplit;
	int rows = (int)ROWS;
	int cols = (int)COLS;

	for (int i = 0; i < rows; i++) {
		getline(in, d);
		dSplit = split(d, delimeter);
		
		for (int j = 0; j < cols; j++) {
			data[i*cols+j] = (float) atof(dSplit[j].c_str());
		}
	}
	return ;
}

// predicate for sorting vector of pair<index,distance> by distance
struct compare
{
	__host__ __device__
	bool operator()(const thrust::pair<int, float> x, const thrust::pair<int, float> y)
	{
		return x.second < y.second;
	}
};

bool compare2 (const std::pair<int, float> x, const std::pair<int, float> y)
{
	return x.second < y.second;
}

int classify(int* nn, thrust::host_vector<int>& labels) {
	int k = (int)K;
	int mid = (k - 1)/2;

	int ones = 0;
	for (int i = 0; i < k; i++)
		ones += labels[nn[i]];

	if (ones > mid) 
		return 1;
	else
		return 0;
	
}



// using cosine distance
__global__ void distances(float *d_records, float *d_distances, int* m)
{
	
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int rows = (int)ROWS;
	int cols = (int)COLS;

	if (i < rows && i != *m){
		float xDotY = 0.0;
		float normX = 0.0;
		float normY = 0.0;
		for (int k = 0; k < cols; k++){
			float x = d_records[i*cols+k];
			float y = d_records[(*m)*cols+k];

			xDotY += (x * y);
			normX += (x * x);
			normY += (y * y);
		}
		float normXY = normX * normY;
		if (normXY == 0)
			d_distances[i] = FLT_MAX;
		else
			d_distances[i] = 1-(xDotY / (normX * normY));
			
	}
	else if (i == *m)
		d_distances[i] = FLT_MAX;
}

int main()
{
	int rows = (int)ROWS;
	int cols = (int)COLS;
	int k = (int)K;

	// connect to file with records
	ifstream rec("PEMS_records01.txt");
	ifstream lab("PEMS_labels01.txt");


	// create corresponding host and device vectors
	thrust::host_vector<float> h_records(rows*cols);
	thrust::host_vector<int> h_labels(rows);
	thrust::host_vector<int> h_nn(k);
	thrust::host_vector<float> h_distances(rows);

	// keep track 
	std::vector<std::pair<int, float> > index_distance(rows);

	// keep track of accurate classification
	int accurate = 0;

	thrust::device_vector<float> d_records(rows*cols);
	thrust::device_vector<float> d_distances(rows);

	// read data from file
	readData1d(lab, h_labels);
	readData(rec, h_records);

	// copy records from host to device
	d_records = h_records;

	// pointers to pass to kernel function
	float *pd_records = thrust::raw_pointer_cast(&d_records[0]);
	float *pd_distances = thrust::raw_pointer_cast(&d_distances[0]);

	// estimate number of blocks given 512 thread per block
	int nThreads = 512;
	int nBlocks = rows/nThreads + 1;

	// variable to specify to kernel what to leave out
	int* xx;
	hipMalloc((void**) &xx, sizeof(int));

	// variable to hold nearest neighbors
	int* nn = (int*) malloc(k*sizeof(int));

	for (int i = 0; i < rows; i++) {
		hipMemcpy(xx, &i, sizeof(int), hipMemcpyHostToDevice);

		distances<<<nBlocks,nThreads>>>(pd_records, pd_distances, xx);

		//synchronize
		hipDeviceSynchronize();

		// copy distances from device to host
		h_distances = d_distances;
		
		// couple distance & index to enable sorting without loss of index info
		for (int m = 0; m < rows; m++)
			index_distance[m] = std::make_pair(m, sqrt(h_distances[m]));

		// sort
		std::sort(index_distance.begin(), index_distance.end(), compare2);

		// retrieve nearest neighbors
		for (int m = 0; m < k; m++)
			nn[m] = index_distance[m].first;

		if (classify(nn, h_labels) == h_labels[i])
			accurate++;
	}


	float accuracy = (accurate*1.0)/((float)ROWS);

	cout << "The accuracy obtained is: " << accuracy << endl;

	lab.close();
	rec.close();
	return 0;
}
